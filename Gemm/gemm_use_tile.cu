#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <hip/hip_runtime.h>

#include "utils.cuh"

__global__ void gemm_use_tile(
    float * __restrict__ a, float * __restrict__ b, float * __restrict__ c,
    const int M, const int N, const int K) {

    // Element number process by thread
    const unsigned kCount = 4;

    // Iteration Count on A
    const unsigned int iterationA = 2;
    
    // Iteration Count on B 
    const unsigned int iterationB = 2;

    // The shape processed by ThreadBlock, 
    unsigned tile_M = blockDim.y * kCount * iterationA;
    unsigned tile_N = blockDim.x * kCount * iterationB;

    // interleave of A and B
    unsigned intervalA = tile_M / iterationA;
    unsigned intervalB = tile_N / iterationB;

    // Top-left element index by this thread
    unsigned int m = blockIdx.y * tile_M + threadIdx.y * kCount;
    unsigned int n = blockIdx.x * tile_N + threadIdx.x * kCount;

    if (m >= M || n >= N)
        return;

    float4 r_a;
    float4 r_b;
    float4 r_c[iterationA][iterationB][4];
    memset(r_c, 0, sizeof(r_c));

    for (int k = 0; k < K; k++) {
#pragma unroll
        for (unsigned iterA = 0; iterA < iterationA; ++iterA) {
            r_a.x = a[OFFSET(m + iterA * intervalA    , k, K)];
            r_a.y = a[OFFSET(m + iterA * intervalA + 1, k, K)];
            r_a.z = a[OFFSET(m + iterA * intervalA+ 2, k, K)];
            r_a.w = a[OFFSET(m + iterA * intervalA+ 3, k, K)];
#pragma unroll
            for (unsigned iterB = 0; iterB < iterationB; ++iterB) {
                r_b = *reinterpret_cast<const float4*>(b + OFFSET(k, n + iterB * intervalB, N));
                r_c[iterA][iterB][0].x += r_a.x * r_b.x;
                r_c[iterA][iterB][0].y += r_a.x * r_b.y;
                r_c[iterA][iterB][0].z += r_a.x * r_b.z;
                r_c[iterA][iterB][0].w += r_a.x * r_b.w;
 
                r_c[iterA][iterB][1].x += r_a.y * r_b.x;
                r_c[iterA][iterB][1].y += r_a.y * r_b.y;
                r_c[iterA][iterB][1].z += r_a.y * r_b.z;
                r_c[iterA][iterB][1].w += r_a.y * r_b.w;

                r_c[iterA][iterB][2].x += r_a.z * r_b.x;
                r_c[iterA][iterB][2].y += r_a.z * r_b.y;
                r_c[iterA][iterB][2].z += r_a.z * r_b.z;
                r_c[iterA][iterB][2].w += r_a.z * r_b.w;

                r_c[iterA][iterB][3].x += r_a.w * r_b.x;
                r_c[iterA][iterB][3].y += r_a.w * r_b.y;
                r_c[iterA][iterB][3].z += r_a.w * r_b.z;
                r_c[iterA][iterB][3].w += r_a.w * r_b.w;           
            }
        }
    }

#pragma unroll
    for (unsigned iterA = 0; iterA < iterationA; ++iterA) {
        for (unsigned iterB = 0; iterB < iterationB; ++iterB) {
            for (unsigned i = 0; i < kCount; i++) {
                *reinterpret_cast<float4*>(c + OFFSET(m + i + iterA * intervalA, n + iterB * intervalB, N)) = r_c[iterA][iterB][i]; 
            }
        }
    }
}	

int main(void) {
    const int BM = 16, BN = 16;
    const int TEST_M = 1024, TEST_N = 1024, TEST_K = 1024;
    //const int BM = 1, BN = 1;
    //const int TEST_M = 16, TEST_N = 16, TEST_K = 16;
    dim3 blockDim_T(BN, BM);
    dim3 gridDim_T((TEST_N / 8 + BN - 1) / BN, (TEST_M / 8 + BM - 1) / BM);

    void (*gpuSgemm) (float*, float*, float*, const int, const int, const int) = gemm_use_tile;
    float max_error = testError(gpuSgemm, gridDim_T, blockDim_T, TEST_M, TEST_N, TEST_K);
    printf("Max error: %f\n", max_error);

    printf("\n Kernel = gemm_use_128\n");
    const int M_list[15] = {128, 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 12288, 16384};
    const int N_list[15] = {128, 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 12288, 16384};
    const int K_list[15] = {1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024};

    const int outer_repeat = 10, inner_repeat = 1;
    const int TESTNUM = 15;
    for (int i = 0; i < TESTNUM; i++) {
        int M = M_list[i];
        int N = N_list[i];
        int K = K_list[i];
        dim3 blockDim(BN, BM);
        dim3 gridDim((N / 4 + BN - 1) / BN, (M / 4 + BM - 1) / BM);
        double max_sec = 0.0;
        double min_sec = DBL_MAX;
        double total_sec = 0.0;

        for (int j = 0; j < outer_repeat; j++) {
            double this_sec = testPerformance(gpuSgemm, gridDim, blockDim, M, N, K, inner_repeat);
            max_sec = max(max_sec, this_sec);
            min_sec = min(min_sec, this_sec);
            total_sec += this_sec;
        }

        double avg_sec = total_sec / outer_repeat;
        double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;
        printf("M N K = %6d %6d %6d, avg_sec = %f, max_sec = %f, min_sec = %f, avg_Gflops = %f\n", M, N, K, avg_sec, max_sec, min_sec, avg_Gflops);
    }
    return 0;
}
