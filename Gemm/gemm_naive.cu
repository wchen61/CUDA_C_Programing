#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <hip/hip_runtime.h>

#include "utils.cuh"

__global__ void naiveSgemm(
    float * __restrict__ a, float * __restrict__ b, float * __restrict__ c,
    const int M, const int N, const int K) {
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    int m = blockIdx.y * blockDim.y + threadIdx.y;
    if (m < M && n < N) {
        float psum = 0.0;
        #pragma unroll
        for (int k = 0; k < K; k++) {
            psum += a[OFFSET(m, k, K)] * b[OFFSET(k, n, N)];
        }
        c[OFFSET(m, n, N)] = psum;
    }
}

int main(void) {
    const int BM = 16, BN = 16;
    const int TEST_M = 1024, TEST_N = 1024, TEST_K = 1024;
    dim3 blockDim_T(BN, BM);
    dim3 gridDim_T((TEST_N + BN - 1) / BN, (TEST_M + BM - 1) / BM);

    void (*gpuSgemm) (float*, float*, float*, const int, const int, const int) = naiveSgemm;
    float max_error = testError(gpuSgemm, gridDim_T, blockDim_T, TEST_M, TEST_N, TEST_K);
    printf("Max error: %f\n", max_error);

    printf("\n Kernel = naiveSgemm\n");
    const int M_list[15] = {128, 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 12288, 16384};
    const int N_list[15] = {128, 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 12288, 16384};
    const int K_list[15] = {1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024};

    const int outer_repeat = 10, inner_repeat = 1;
    const int TESTNUM = 15;
    for (int i = 0; i < TESTNUM; i++) {
        int M = M_list[i];
        int N = N_list[i];
        int K = K_list[i];
        dim3 blockDim(BN, BM);
        dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);
        double max_sec = 0.0;
        double min_sec = DBL_MAX;
        double total_sec = 0.0;

        for (int j = 0; j < outer_repeat; j++) {
            double this_sec = testPerformance(gpuSgemm, gridDim, blockDim, M, N, K, inner_repeat);
            max_sec = max(max_sec, this_sec);
            min_sec = min(min_sec, this_sec);
            total_sec += this_sec;
        }

        double avg_sec = total_sec / outer_repeat;
        double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;
        printf("M N K = %6d %6d %6d, avg_sec = %f, max_sec = %f, min_sec = %f, avg_Gflops = %f\n", M, N, K, avg_sec, max_sec, min_sec, avg_Gflops);
    }
    return 0;
}
