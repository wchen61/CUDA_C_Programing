#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../common.h"


__global__ void nestedHelloWorld(int const iSize, int iDepth) {
    int tid = threadIdx.x;
    printf("Recursion=%d: Hello World from thread %d block %d\n", iDepth, tid, blockIdx.x);

    if (iSize == 1) return;

    int nThreads = iSize >> 1;
    if (tid == 0 && nThreads > 0) {
        nestedHelloWorld<<<1, nThreads>>>(nThreads, ++iDepth);
        printf("-------> nested execution depth: %d\n", iDepth);
    }
}

int main(int argc, char **argv) {
    int size = 8;
    int blockSize = 8;
    int igrid = 1;

    if (argc > 1) {
        igrid = atoi(argv[1]);
        size = igrid * blockSize;
    }

    dim3 block(blockSize, 1);
    dim3 grid((size + block.x - 1) / block.x, 1);

    printf("%s Execution Configuration: grid %d block %d\n", argv[0], grid.x, block.x);

    nestedHelloWorld<<<grid, block>>>(block.x, 0);

    CHECK(hipGetLastError());
    CHECK(hipDeviceReset());
    return 0;
}