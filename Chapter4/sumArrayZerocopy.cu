#include "hip/hip_runtime.h"
#include "../common.h"


/*void initialData(float *ip, int size) {
    time_t t;
    srand((unsigned int) time(&t));
    for (int i=0; i<size; i++) {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

bool checkResult(float *A, float *B, int size) {
    double epsilon = 1.0E-8;
    for (int idx = 0; idx < size; idx++) {
        if (abs(A[idx] - B[idx]) > epsilon) {
            return false;
        }
        //printf("%d : %f %f\n", idx, A[idx], B[idx]);
    }
    return true;
}*/

void sumArraysOnHost(float *A, float *B, float *C, const int N) {
    for (int idx = 0; idx < N; idx++) {
        C[idx] = A[idx] + B[idx];
    }
}

__global__ void sumArrays(float *A, float *B, float*C, int nElem) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nElem)
        C[idx] = A[idx] + B[idx];
}

__global__ void sumArraysZeroCopy(float *A, float *B, float*C, int nElem) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nElem)
        C[idx] = A[idx] + B[idx];
}

int main(int argc, char **argv) {
    int dev = 0;
    hipSetDevice(dev);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    if (!deviceProp.canMapHostMemory) {
        printf("Device %d does not support mapping CPU host memory!\n", dev);
        hipDeviceReset();
        exit(EXIT_SUCCESS);
    }
    printf("Using Device %d : %s\n", dev, deviceProp.name);

    int ipower = 10;
    if (argc > 1) ipower = atoi(argv[1]);
    int nElem = 1 << ipower;
    size_t nBytes = nElem * sizeof(float);
    if (ipower < 18) {
        printf("Vector size %d power %d nbytes %3.0f KB\n", nElem, ipower, (float)nBytes / (1024.0f));
    } else {
        printf("Vector size %d power %d nbytes %3.0f MB\n", nElem, ipower, (float)nBytes / (1024.0f * 1024.0f));
    }

    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float*)malloc(nBytes);
    h_B = (float*)malloc(nBytes);
    hostRef = (float*)malloc(nBytes);
    gpuRef = (float*)malloc(nBytes);

    initialData(h_A, nElem);
    initialData(h_B, nElem);
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    sumArraysOnHost(h_A, h_B, hostRef, nElem);

    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);

    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    int iLen = 512;
    dim3 block(iLen);
    dim3 grid((nElem + block.x - 1) / block.x);

    sumArrays<<<grid, block>>>(d_A, d_B, d_C, nElem);

    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    checkResult(hostRef, gpuRef, nElem);

    hipFree(d_A);
    hipFree(d_B);
    free(h_A);
    free(h_B);

    unsigned int flags = hipHostMallocMapped;
    hipHostAlloc((void**)&h_A, nBytes, flags);
    hipHostAlloc((void**)&h_B, nBytes, flags);
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    //hipHostGetDevicePointer((void**)&d_A, (void*)h_A, 0);
    //hipHostGetDevicePointer((void**)&d_B, (void*)h_B, 0);

    sumArraysOnHost(h_A, h_B, hostRef, nElem);

    //sumArraysZeroCopy<<<grid, block>>>(d_A, d_B, d_C, nElem);
    sumArraysZeroCopy<<<grid, block>>>(h_A, h_B, d_C, nElem);
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    checkResult(hostRef, gpuRef, nElem);
    hipFree(d_C);
    hipHostFree(d_A);
    hipHostFree(d_B);

    free(hostRef);
    free(gpuRef);

    hipDeviceReset();
    return EXIT_SUCCESS;
}