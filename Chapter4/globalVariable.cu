#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ float devData;

__global__ void checkGlobalVariable() {
    printf("Device: the value of the global variable is %f\n", devData);
    devData += 2.0f;
}

int main(void) {
    float value = 3.14f;
    hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float));
    printf("Host: cpoed %f to the global variable\n", value);

    checkGlobalVariable<<<1, 1>>>();

    //cudaMemcpyFromSymbol(&value, devData, sizeof(float));

    float *dptr = NULL;
    hipGetSymbolAddress((void**)&dptr, HIP_SYMBOL(devData));
    hipMemcpy(&value, dptr, sizeof(float), hipMemcpyDeviceToHost);
    printf("Host: the value changed by the kernel to %f\n", value);
    
    hipDeviceReset();
    return EXIT_SUCCESS;
}