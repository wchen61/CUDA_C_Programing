#include "hip/hip_runtime.h"
#include "../common.h"
#include <hip/hip_runtime.h>

#define BDIMX 32 
#define BDIMY 16 
#define IPAD 2

void printData(char* msg, int *in, const int size) {
    printf("%s: ", msg);
    for (int i=0; i<size; i++) {
        printf("%5d", in[i]);
        fflush(stdout);
    }
    printf("\n");
    return;
}

__global__ void setRowReadRow (int *out) {
    __shared__ int tile[BDIMY][BDIMX];
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    tile[threadIdx.y][threadIdx.x] = idx;
    __syncthreads();
    out[idx] = tile[threadIdx.y][threadIdx.x];
}

__global__ void setColReadCol(int *out) {
    __shared__ int tile[BDIMX][BDIMY];
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    tile[threadIdx.x][threadIdx.y] = idx;
    __syncthreads();
    out[idx] = tile[threadIdx.x][threadIdx.y];
}

__global__ void setRowReadCol(int *out) {
    __shared__ int tile[BDIMY][BDIMX];
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int irow = idx / blockDim.y;
    unsigned int icol = idx % blockDim.y;

    tile[threadIdx.y][threadIdx.x] = idx;

    __syncthreads();
    out[idx] = tile[icol][irow];
}

__global__ void setRowReadColDyn(int *out) {
    extern __shared__ int tile[];
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int irow = idx / blockDim.y;
    unsigned int icol = idx % blockDim.y;

    unsigned int col_idx = icol * blockDim.x + irow;

    tile[idx] = idx;

    __syncthreads();

    out[idx] = tile[col_idx];
}

__global__ void setRowReadColPad(int *out) {
    __shared__ int tile[BDIMY][BDIMX + IPAD];
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int irow = idx / blockDim.y;
    unsigned int icol = idx % blockDim.y;
    tile[threadIdx.y][threadIdx.x] = idx;
    __syncthreads();

    out[idx] = tile[icol][irow];
}

__global__ void setRowReadColDynPad(int *out) {
    extern __shared__ int tile[];
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int irow = idx / blockDim.y;
    unsigned int icol = idx % blockDim.y;

    unsigned int row_idx = threadIdx.y * (blockDim.x + IPAD) + threadIdx.x;
    unsigned int col_idx = icol * (blockDim.x + IPAD) + irow;

    tile[row_idx] = idx;
    __syncthreads();

    out[idx] = tile[col_idx];
}

int main(int argc, char** argv) {
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s at ", argv[0]);
    printf("device %d: %s", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    hipSharedMemConfig pConfig;
    CHECK(hipDeviceGetSharedMemConfig(&pConfig));
    printf(" with Bank Mode : %s", pConfig == 1 ? "4-Bytes" : "8-Bytes");

    int nx = BDIMX;
    int ny = BDIMY;
    bool iprintf = 0;

    if (argc > 1) iprintf = atoi(argv[1]);
    size_t nBytes = nx * ny * sizeof(int);

    dim3 block(BDIMX, BDIMY);
    dim3 grid(1, 1);

    printf(" <<<grid (%d, %d) block (%d, %d)>>>\n", grid.x, grid.y, block.x, block.y);

    int* d_out;
    int* gpuRef;
    CHECK(hipMalloc((int**)&d_out, nBytes));

    gpuRef = (int*)malloc(nBytes);

    CHECK(hipMemset(d_out, 0, nBytes));
    setRowReadRow<<<grid, block>>>(d_out);
    CHECK(hipMemcpy(gpuRef, d_out, nBytes, hipMemcpyDeviceToHost));
    if (iprintf) printData("setRowReadRow\t\t\t", gpuRef, nx, ny);

    CHECK(hipMemset(d_out, 0, nBytes));
    setColReadCol<<<grid, block>>>(d_out);
    CHECK(hipMemcpy(gpuRef, d_out, nBytes, hipMemcpyDeviceToHost));
    if (iprintf) printData("setColReadCol\t\t\t", gpuRef, nx, ny);

    CHECK(hipMemset(d_out, 0, nBytes));
    setRowReadCol<<<grid, block>>>(d_out);
    CHECK(hipMemcpy(gpuRef, d_out, nBytes, hipMemcpyDeviceToHost));
    if (iprintf) printData("setRowReadCol\t\t\t", gpuRef, ny, nx);

    CHECK(hipMemset(d_out, 0, nBytes));
    setRowReadColDyn<<<grid, block, BDIMY * BDIMX * sizeof(int)>>>(d_out);
    CHECK(hipMemcpy(gpuRef, d_out, nBytes, hipMemcpyDeviceToHost));
    if (iprintf) printData("setRowReadColDyn\t\t\t", gpuRef, ny, nx);

    CHECK(hipMemset(d_out, 0, nBytes));
    setRowReadColPad<<<grid, block>>>(d_out);
    CHECK(hipMemcpy(gpuRef, d_out, nBytes, hipMemcpyDeviceToHost));
    if (iprintf) printData("setRowReadColPad\t\t\t", gpuRef, ny, nx);

    CHECK(hipMemset(d_out, 0, nBytes));
    setRowReadColDynPad<<<grid, block, BDIMY * (BDIMX + IPAD) * sizeof(int)>>>(d_out);
    CHECK(hipMemcpy(gpuRef, d_out, nBytes, hipMemcpyDeviceToHost));
    if (iprintf) printData("setRowReadColDynPad\t\t\t", gpuRef, ny, nx);
}

