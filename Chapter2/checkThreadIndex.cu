#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHECK(call)                                                              \
{                                                                                \
    const hipError_t error = call;                                              \
    if (error != hipSuccess) {                                                  \
        printf("Error: %s : %d,", __FILE__, __LINE__);                           \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error));       \
        exit(1);                                                                 \
    }                                                                            \
}

void initialInt(int *ip, int size) {
    for (int i=0; i<size; i++) {
        ip[i] = i;
    }
}

void printMatrix(int *C, const int nx, const int ny) {
    int *ic = C;
    printf("\nMatrix: (%d, %d)\n", nx, ny);
    for (int iy = 0; iy < ny; iy++) {
        for (int ix = 0; ix < nx; ix++) {
            printf("%3d", ic[ix]);
        }
        ic += nx;
        printf("\n");
    }
    printf("\n");
}

__global__ void printThreadIndx(int *A, const int nx, const int ny) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    printf("threadIdx(%d, %d), blockIdx(%d, %d), corrdinate (%d, %d) "
           "global index %2d ival %2d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y,
           ix, iy, idx, A[idx]);
}

int main(int argc, char **argv) {
    printf("%s Starting...\n", argv[0]);

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    int nx = 8;
    int ny = 6;
    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);

    int *h_A;
    h_A = (int*)malloc(nBytes);
    initialInt(h_A, nxy);
    printMatrix(h_A, nx, ny);

    int *d_MatA;
    hipMalloc((void**)&d_MatA, nBytes);
    hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);

    dim3 block(4, 2);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    printThreadIndx<<<grid, block>>>(d_MatA, nx, ny);
    hipDeviceSynchronize();

    hipFree(d_MatA);
    free(h_A);

    hipDeviceReset();
    return 0;
}