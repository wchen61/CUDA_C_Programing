
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <stdio.h>

#define CHECK(call)                                                              \
{                                                                                \
    const hipError_t error = call;                                              \
    if (error != hipSuccess) {                                                  \
        printf("Error: %s : %d,", __FILE__, __LINE__);                           \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error));       \
        exit(1);                                                                 \
    }                                                                            \
}

void sumArraysOnHost(float *A, float *B, float *C, const int N) {
    for (int idx = 0; idx < N; idx++) {
        C[idx] = A[idx] + B[idx];
    }
}

bool checkResult(float *A, float *B, int size) {
    double epsilon = 1.0E-8;
    for (int idx = 0; idx < size; idx++) {
        if (abs(A[idx] - B[idx]) > epsilon) {
            return false;
        }
        //printf("%d : %f %f\n", idx, A[idx], B[idx]);
    }
    return true;
}

__global__ void sumArraysOnDevice(float *A, float *B, float*C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    C[idx] = A[idx] + B[idx];
}

void initialData(float *ip, int size) {
    time_t t;
    srand((unsigned int) time(&t));
    for (int i=0; i<size; i++) {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}



int main(int argc, char **argv) {
    int nElem = 1024;
    size_t nBytes = nElem * sizeof(float);
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    float *h_C1;

    h_A = (float*)malloc(nBytes);
    h_B = (float*)malloc(nBytes);
    h_C = (float*)malloc(nBytes);
    h_C1 = (float*)malloc(nBytes);
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);

    initialData(h_A, nElem);
    initialData(h_B, nElem);
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    sumArraysOnHost(h_A, h_B, h_C, nElem);

    sumArraysOnDevice<<<1, nElem>>>(d_A, d_B, d_C);

    hipMemcpy(h_C1, d_C, nBytes, hipMemcpyDeviceToHost);

    if (!checkResult(h_C, h_C1, nElem)) {
        printf("Result is not identity!\n");
    } else {
        printf("Result is identity!\n");
    }

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C1);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}